#include "hip/hip_runtime.h"
#include "hipblas.h"
#include <stdio.h>
#include <iostream>
#include "numer_cublas_wrappers.h"

template<>
hipblasStatus_t BlasWrapper<float>::cublasGemm(hipblasHandle_t handle, hipblasOperation_t transa, hipblasOperation_t transb, 
						   int m, int n, int k,
                           const float           *alpha,
                           const float           *A, int lda,
                           const float           *B, int ldb,
                           const float           *beta,
                           float           *C, int ldc){
  //std::cout<<"blas wrapper: float"<<std::endl;

	return hipblasSgemm(handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
};

template<>
hipblasStatus_t BlasWrapper<double>::cublasGemm(hipblasHandle_t handle, hipblasOperation_t transa, hipblasOperation_t transb, 
						   int m, int n, int k,
                           const double           *alpha,
                           const double           *A, int lda,
                           const double           *B, int ldb,
                           const double           *beta,
                           double           *C, int ldc){

  //std::cout<<"blas wrapper: double"<<std::endl;

	return hipblasDgemm(handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
};

/*template<typename T>
hipblasStatus_t BlasWrapper<T>::cublasGemv(hipblasHandle_t handle, hipblasOperation_t trans,
                           int m, int n,
                           const T           *alpha,
                           const T           *A, int lda,
                           const T           *x, int incx,
                           const T           *beta,
                           T           *y, int incy){
	return hipblasSgemv( handle,  trans, m,  n, alpha, A, lda, x, incx, beta, y, incy);
};
*/
template<>
hipblasStatus_t BlasWrapper<float>::cublasGemv(hipblasHandle_t handle, hipblasOperation_t trans,
                           int m, int n,
                           const float           *alpha,
                           const float           *A, int lda,
                           const float           *x, int incx,
                           const float           *beta,
                           float           *y, int incy){
	return hipblasSgemv( handle,  trans, m,  n, alpha, A, lda, x, incx, beta, y, incy);
};

template<>
hipblasStatus_t BlasWrapper<double>::cublasGemv(hipblasHandle_t handle, hipblasOperation_t trans,
                           int m, int n,
                           const double           *alpha,
                           const double           *A, int lda,
                           const double           *x, int incx,
                           const double           *beta,
                           double           *y, int incy){
	return hipblasDgemv( handle,  trans, m,  n, alpha, A, lda, x, incx, beta, y, incy);
};


/*template<typename T>
hipblasStatus_t BlasWrapper<T>::cublasGeam(hipblasHandle_t handle,
                          hipblasOperation_t transa, hipblasOperation_t transb,
                          int m, int n,
                          const T           *alpha,
                          const T           *A, int lda,
                          const T           *beta,
                          const T           *B, int ldb,
                          T           *C, int ldc){
	return hipblasSgeam(handle, transa, transb, m, n, alpha, A, lda, beta, B,  ldb, C, ldc);

};
*/
template<>
hipblasStatus_t BlasWrapper<float>::cublasGeam(hipblasHandle_t handle,
                          hipblasOperation_t transa, hipblasOperation_t transb,
                          int m, int n,
                          const float           *alpha,
                          const float           *A, int lda,
                          const float           *beta,
                          const float           *B, int ldb,
                          float           *C, int ldc){
	return hipblasSgeam(handle, transa, transb, m, n, alpha, A, lda, beta, B,  ldb, C, ldc);
};

template<>
hipblasStatus_t BlasWrapper<double>::cublasGeam(hipblasHandle_t handle,
                          hipblasOperation_t transa, hipblasOperation_t transb,
                          int m, int n,
                          const double           *alpha,
                          const double           *A, int lda,
                          const double           *beta,
                          const double           *B, int ldb,
                          double           *C, int ldc){

	return hipblasDgeam(handle, transa, transb, m, n, alpha, A, lda, beta, B,  ldb, C, ldc);
};

template class BlasWrapper<float>;
template class BlasWrapper<double>;
