#include <stdio.h>
#include <iostream>
#include "numer_buffer.h"


//from row,col to index, for a given leading dimension
#define IDX2C(i,j,ld) (((j)*(ld))+(i))

//from row major to column major indexes
#define IDX2RRM(idx,C) (idx/C) //Index to row for RM matrix
#define IDX2CRM(idx,C) (idx%C) //index to col for RM matrix

// from column major to row major indexes
#define IDX2RCM(idx,R) (idx%R) //Idex to row for CM matrix
#define IDX2CCM(idx,R) (idx/R) //index to col for CM matrix

template<typename T>
NumerMatrixFloatBuffer<T>::NumerMatrixFloatBuffer():NumerFloatBuffer<T>() {
    this->_rows = 1;
    this->_cols = h_data->size();
}

template<typename T>
NumerMatrixFloatBuffer<T>::NumerMatrixFloatBuffer(unsigned int rows, unsigned int cols, MatrixOrientation orientation):NumerFloatBuffer<T>(rows*cols){
    this->_rows = rows;
    this->_cols = cols;
    this->orientation = orientation;
}

template<typename T>
NumerMatrixFloatBuffer<T>::~NumerMatrixFloatBuffer() {
}

template<typename T>
unsigned int NumerMatrixFloatBuffer<T>::size() {
    return this->h_data->size();
}

template<typename T>
void NumerMatrixFloatBuffer<T>::write(ErlNifEnv *env, ERL_NIF_TERM data) {
    ERL_NIF_TERM head_row;
    ERL_NIF_TERM head;
    double value;
    long lvalue;

    unsigned ld = this->_rows; //number of rows; this is the lead dimension (ld) in row major matrices
    unsigned C = this->_cols; 
    unsigned long idx = 0;
   
    //CUBLAS uses column major matrices. this converts Erlang row major matrix ("list-of-lists") to column major one dimensional vector   
    if(this->orientation == ROW_MAJOR){
        while (enif_get_list_cell(env, data, &head_row, &data)) {
          while (enif_get_list_cell(env, head_row, &head, &head_row))
            if (enif_get_double(env, head, &value)) {
                //this->data->at((IDX2C(IDX2RRM(idx,C), IDX2CRM(idx,C), ld)) ) = value;
                (*(this->h_data))[IDX2C(IDX2RRM(idx,C), IDX2CRM(idx,C), ld)]  = (T)value;
                ++idx;
            }else if (enif_get_long(env, head, &lvalue)) {
                //this->data->at(IDX2C(IDX2RRM(idx,C), IDX2CRM(idx,C), ld)) = (double)lvalue;
                (*(this->h_data))[IDX2C(IDX2RRM(idx,C), IDX2CRM(idx,C), ld)] = (T)lvalue;
                ++idx;
            }
        }
        
        if(idx != this->h_data->size()){ 
            throw std::runtime_error("ERROR: Data does not fit the matrix size.");
        }
    } else {
        while (enif_get_list_cell(env, data, &head, &data)) {
            if (enif_get_double(env, head, &value)) {
                this->h_data->push_back(value);
            }else if (enif_get_long(env, head, &lvalue)) {
                this->h_data->push_back((T)lvalue);
            }
        }
    }

    d_data->resize(h_data->size());
    thrust::copy(this->h_data->begin(), this->h_data->end(), this->d_data->begin());
    //*d_data = *h_data;
    hipDeviceSynchronize();
}

template<typename T>
ERL_NIF_TERM NumerMatrixFloatBuffer<T>::toErlTerms(ErlNifEnv *env) {
    typename std::vector<T>::iterator iter;
    ERL_NIF_TERM retval = enif_make_list(env, 0);
    ERL_NIF_TERM row;

    unsigned R = this->_rows;
    unsigned Ridx;
    unsigned long idx = 0;
    std::vector<ERL_NIF_TERM> rows;
    

    h_data->clear();
    h_data->resize(d_data->size());
    thrust::copy(d_data->begin(), d_data->end(), h_data->begin());


    if(this->rows() > 1 && this->orientation == ROW_MAJOR){
        for(int i=0; i<this->_rows; i++){
            row = enif_make_list(env, 0);
            rows.push_back(row);
        }
        
        for (iter = h_data->end(); iter != h_data->begin(); ) {
                --iter;
                Ridx = IDX2RCM(idx,R);
                rows[Ridx] = enif_make_list_cell(env, enif_make_double(env, (double)*iter), rows[Ridx]);
                ++idx;
        };
        for(int i=0; i<this->_rows; i++){
            retval = enif_make_list_cell(env, rows[i], retval);
        };
    }else{
        if (this->h_data->size() > 0) {
            for (iter = h_data->end(); iter != h_data->begin();) {
                --iter;
                retval = enif_make_list_cell(env, enif_make_double(env, (double)*iter), retval);
            }
        }        
    }

    return retval;
}

template<typename T>
void NumerMatrixFloatBuffer<T>::clear() {
    this->h_data->clear();
    this->d_data->clear();
    //this->_size = 0;
}

template class NumerMatrixFloatBuffer<float>;
template class NumerMatrixFloatBuffer<double>;
